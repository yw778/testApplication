#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include "mbgd_1.h"
#include "spamfilter_utils_cuda.cuh"

/* Parallel approach to batch gradient descent using multiple "mini-batches"
rather than the whole batch. Each block is assigned to one mini-batch with
multiple threads assigned to each data point. */


// pointers to device global variables
static FeatureType *d_parameter_vector, *d_data_points;
static LabelType *d_labels;

// Allocate space for the data set, labels and parameter vector in global memory
// Then, copy values for those host variables to the device variables
static void setCudaVariables(
    size_t num_features,
    size_t num_data_points,
    FeatureType* data_points,
    LabelType* labels,
    FeatureType* parameter_vector) {

    checkCudaErrors(hipMalloc(&d_parameter_vector, num_features 
                                * sizeof(FeatureType)));
    checkCudaErrors(hipMalloc(&d_data_points, num_data_points * num_features 
                                * sizeof(FeatureType)));
    checkCudaErrors(hipMalloc(&d_labels, num_data_points * sizeof(LabelType)));

    checkCudaErrors(hipMemcpy(d_data_points, data_points, num_data_points
                                * num_features * sizeof(FeatureType),
                                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_labels, labels, num_data_points
                                * sizeof(LabelType), 
                                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_parameter_vector, parameter_vector,
                                num_features * sizeof(FeatureType), 
                                    hipMemcpyHostToDevice));
}


static void cleanUp() {

    checkCudaErrors(hipFree(d_parameter_vector));
    checkCudaErrors(hipFree(d_data_points));
    checkCudaErrors(hipFree(d_labels));
}


static __device__ void d_partialDotProduct(
    FeatureType* data_point_i,
    FeatureType* parameter_vector,
    FeatureType* shared_memory,
    size_t num_features,
    size_t threads_per_datapoint) {
    
    FeatureType partial_dot = 0;

    size_t thread_offset = threadIdx.x % threads_per_datapoint;

    // strided sum of element-wise products
    for (size_t j = thread_offset; j < num_features; j += threads_per_datapoint) {
        partial_dot += data_point_i[j] * parameter_vector[j];
    }

    // result of the partial dot product is stored in shared memory
    shared_memory[threadIdx.x] = partial_dot;
}


// computes gradient for a mini-batch of size batch_size in the training_set
// from starting_point
static __device__ void d_gradientForMiniBatch(
    FeatureType* data_points,
    FeatureType* parameter_vector,
    FeatureType* labels,
    size_t num_features,
    size_t num_data_points,
    size_t batch_size,
    size_t threads_per_datapoint,
    FeatureType* gradient){

    // array probabilities_of_positive in shared_memory of size batch_size
    float *probabilities_of_positive = (float*)&gradient[num_features];
    // array dot_product in shared_memory of size threads_per_datapoint * batch_size
    float *dot_product = (float*)&probabilities_of_positive[batch_size];
    
    size_t tidx = threadIdx.x;
    size_t bidx = blockIdx.x;
    size_t point_idx = bidx * batch_size + tidx / threads_per_datapoint;
    // thread index relative to data point
    size_t relative_tidx = threadIdx.x % threads_per_datapoint; 
    // computes logistic function for each data point in the mini batch
    // size_t starting_point = point_idx * num_features;
    if (point_idx < num_data_points){
        d_partialDotProduct( &data_points[point_idx * num_features], 
                                parameter_vector,
                                dot_product, num_features, 
                                threads_per_datapoint );
    }
    __syncthreads();

    for (size_t s = threads_per_datapoint / 2; s > 0; s>>=1) {
        if (relative_tidx < s) {
            dot_product[tidx] += dot_product[tidx + s];
        }
    }

    __syncthreads();

    // Use only batch_size number of threads to do the following computations
    if (tidx < batch_size){
        // index of summed dot product value
        size_t didx  = tidx * threads_per_datapoint;
        probabilities_of_positive[tidx] = d_logisticFunction(dot_product[didx])
                - labels[bidx * batch_size + tidx];
    }
     
    
    __syncthreads();

    //Finishes computation of gradient
    size_t threads_per_mini_batch = batch_size * threads_per_datapoint;
    float factor = 1.0f/batch_size;
    d_matrixVectorMultiply( data_points,
                            probabilities_of_positive,
                            factor,
                            batch_size,
                            num_features,
                            threads_per_mini_batch,
                            gradient );
}


static __global__ void p_MiniBatchGradientDescent(
    FeatureType* data_points,
    FeatureType* parameter_vector,
    LabelType* labels,
    size_t num_features,
    size_t num_data_points,
    size_t batch_size,
    size_t threads_per_datapoint,
    double step_size) {

    extern __shared__ FeatureType shared_memory[];
    FeatureType *gradient = shared_memory;
    size_t threads_per_mini_batch = threads_per_datapoint * batch_size;
    
    // set all gradient values to 0
    d_memset(gradient, 0, num_features, threads_per_mini_batch);
    // Finds gradient for mini-batch
    d_gradientForMiniBatch( data_points,
                            parameter_vector,
                            labels,
                            num_features,
                            num_data_points,
                            batch_size,
                            threads_per_datapoint,
                            gradient );

    __syncthreads();

    // Updates the parameters
    d_updateParameters( gradient, parameter_vector, num_features,
                        threads_per_mini_batch, step_size );
}


void trainParallelMiniBatchGradientDescent( 
    DataSet training_set,
    TrainingOptions training_options ) {

    // shuffle data points
    /* shuffleKeyValue( training_set.data_points, training_set.labels, 
                     training_set.num_data_points, training_set.num_features ); */

    setCudaVariables( training_set.num_features,
                      training_set.num_data_points,
                      training_set.data_points, 
                      training_set.labels, 
                      training_set.parameter_vector );
    
    // read configuration parameters and initialize grid and block dimensions
    double step_size = *training_options.step_size;

    const double threads_per_datapoint =
            (fieldExists(training_options.config_params, "threads_per_datapoint"))
            ? training_options.config_params["threads_per_datapoint"]
            : THREADS_PER_DATAPOINT;

    const double batch_size =
            (fieldExists(training_options.config_params, "batch_size"))
            ? training_options.config_params["batch_size"]
            : BATCH_SIZE;

    const double characteristic_time =
            (fieldExists(training_options.config_params, "characteristic_time"))
            ? training_options.config_params["characteristic_time"]
            : CHARACTERISTIC_TIME;

    size_t curr_num_epochs =
            (fieldExists(training_options.config_params, "curr_num_epochs"))
            ? training_options.config_params["curr_num_epochs"]
            : 0;

    double annealed_step_size = step_size;

    const dim3 block_size(threads_per_datapoint * batch_size, 1, 1);
    size_t num_blocks = DIVIDE_AND_CEIL( training_set.num_data_points,
                                            batch_size );
    const dim3 grid_size(num_blocks, 1, 1);

    FeatureType threads_per_batch = threads_per_datapoint * batch_size;
    const size_t shared_memory_size = batch_size * sizeof(float) 
            + (threads_per_batch) * sizeof(FeatureType) + training_set.num_features
            * sizeof(FeatureType);
 
    if (checkDeviceProps(shared_memory_size, block_size, grid_size)) {
        // iterate if dimensions are okay
        for (size_t k = 0; k < training_options.num_epochs; k++) {
        annealed_step_size = training_options.config_params["initial_step_size"]
                                / (1.0
                                    + (curr_num_epochs
                                       * training_set.num_data_points
                                       / characteristic_time));
            curr_num_epochs++;

            // adjust step size with a modified version of simulated annealing

            // call kernel and check for errors
            p_MiniBatchGradientDescent
                    <<<grid_size, block_size, shared_memory_size>>>(
                        d_data_points, 
                            d_parameter_vector,
                            d_labels,
                            training_set.num_features, 
                            training_set.num_data_points, 
                            batch_size,
                            threads_per_datapoint,
                            annealed_step_size );
            hipDeviceSynchronize();
            checkCudaErrors(hipGetLastError());
        }
        checkCudaErrors(hipMemcpy( training_set.parameter_vector, 
                                    d_parameter_vector,
                                    training_set.num_features 
                                    * sizeof(FeatureType), 
                                    hipMemcpyDeviceToHost));
    }

    *training_options.step_size = annealed_step_size; 

    cleanUp();
}
