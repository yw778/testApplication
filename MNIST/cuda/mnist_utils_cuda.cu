#include "hip/hip_runtime.h"
#include <math.h>

#include "mnist_utils_cuda.cuh"
#include "utils/mnist_timer.h"



//------cublas function for softmax-------
void p_softmaxFunction(hipblasHandle_t handle, 
    FeatureType* d_theta, FeatureType* d_x_i,
    FeatureType* posibilities_positive,
    const size_t num_feats, const size_t num_labels) {

    float alf=1.0;
    float beta=0;
    // refer to http://stackoverflow.com/questions/21164373/the-cublas-function-call-cublassgemv
    hipblasSgemv(handle, HIPBLAS_OP_T, num_feats, num_labels,
         &alf, d_theta, num_feats, d_x_i, 1, 
         &beta, posibilities_positive, 1);

    hipDeviceSynchronize();

    float sum = 0;


    for(size_t i=0 ; i< num_labels; i++){
        posibilities_positive[i] = exp(posibilities_positive[i]);
        sum += posibilities_positive[i];
    } 

    for(size_t i=0 ; i< num_labels; i++){
        posibilities_positive[i] /= sum;
    } 
}

void p_softmaxFunction2(hipblasHandle_t handle, 
    FeatureType* d_theta, FeatureType* d_x_i,
    FeatureType* posibilities_positive,
    const size_t num_feats, 
    const size_t num_labels){

    for(size_t i=0;i< num_labels;i++){
        
        posibilities_positive[i] = p_dot_product(handle,
            d_theta[i*num_feats],
            d_x_i,
            num_feats);
    }

    float sum = 0;

    for(size_t i=0 ; i< num_labels; i++){
        posibilities_positive[i] = exp(posibilities_positive[i]);
        sum += posibilities_positive[i];
    } 

    for(size_t i=0 ; i< num_labels; i++){
        posibilities_positive[i] /= sum;
    } 

}

// adds two device vectors with CuBLAS and stores the results in the first one
void p_add_vectors(hipblasHandle_t handle, float* a, float* b, const size_t size, const float scale_for_a) {
    hipblasSaxpy(handle, size, &scale_for_a, b, 1, a, 1);
}

// update parameter in parallel in CuBLAS
void p_updateParameters(hipblasHandle_t handle, FeatureType* d_theta, FeatureType* d_gradient, size_t num_feats, float step_size, bool revert) {
    float sign = revert ? 1 : -1;
    step_size *= sign;
    hipblasSaxpy(handle, num_feats, &step_size, d_gradient, 1, d_theta, 1);
}


float p_dot_product(hipblasHandle_t handle, float* d_a, float* d_b, const size_t num_elems) {

    float result[1];
    hipblasSdot (handle, num_elems, d_a, 1, d_b, 1, result);
    hipDeviceSynchronize();
    return *result;
}


// adds two device vectors with CuBLAS and stores the results in the first one
// void p_addVectors(float* a, float* b, const size_t size, const float scale_for_b) {
     // hipblasSaxpy(handle, size, &scale_for_a, b, 1, a, 1);
//     }
// }


// void addVectors(float* a, float* b, const size_t size, const float scale_for_b) {
//     for (size_t j = 0; j < size; j++) {
//         a[j] += scale_for_b * b[j];
//     }
// }

// computes dot product with CuBLAS for two given vectors a and b
// float p_dotProduct(float* d_a, float* d_b, const size_t num_elems) {

     // float result[1];
     // hipblasSdot (handle, num_elems, d_a, 1, d_b, 1, result);
     // hipDeviceSynchronize();
     // return *result;
 // }

// float dotProduct(float* d_a, float* d_b, const size_t num_elems) {
//      float result = 0;
//      for (size_t j = 0; j < num_elems; j++) {
//          result += d_a[j] * d_b[j];
//      }
//      return result;
//  }


// Parallel implementation of matrix vector multiplication. Each thread goes
// a certain number of features and strides by the number of threads in the 
// whole mini batch.
__device__ void d_matrixVectorMultiply(
    FeatureType* matrix,
    FeatureType* vect,
    float scalar,
    size_t batch_size,
    size_t num_features,
    size_t threads_per_mini_batch,
    FeatureType* result) {

    size_t tidx = threadIdx.x;
    size_t bidx = blockIdx.x;
    for (int j = 0; j < batch_size; j++) {
        for (int i = tidx; i < num_features; i += threads_per_mini_batch) {
            // index of the point with respect to the whole dataset
            size_t point_idx = bidx * batch_size + j;
            // index of the feature with respect to all features in the dataset
            size_t feature_idx = point_idx * num_features + i;
            result[i] += matrix[feature_idx] * vect[j] * scalar;
        }
    }
}

// Grabdient = probility_matrix_transpose * datapoint_matrix
// version 2 more faster but thraeds number is limited
__device__ void d_matrixMatrixMultiply2(
    FeatureType* datapoint_matrix,
    FeatureType* probility_matrix,
    float scalar,
    size_t batch_size,
    size_t num_features,
    size_t threads_per_mini_batch,
    FeatureType* result) {

    size_t tidx = threadIdx.x;
    size_t bidx = blockIdx.x;

    // size_t thread_offset = threadIdx.x % threads_per_datapoint;
    size_t num_thread_each_label = threads_per_mini_batch / LABEL_CLASS;
    //index relative to each label(corresponding to 784 parameter) 
    //Eg: 320 thread for 10 label -> each label 32 thread
    size_t tidx_label =  tidx / num_thread_each_label;
    size_t relative_tidx_label =  tidx % num_thread_each_label;
 
    for (int j = 0; j < batch_size; j++) {
        for (int i = relative_tidx_label; i < num_features; i += num_thread_each_label) {
            // index of the point with respect to the whole dataset
            size_t point_idx = bidx * batch_size + j;
            // index of the feature with respect to all features in the dataset
            size_t feature_idx = point_idx * num_features + i;
            //gradient result 
            result[i+tidx_label*num_features] += datapoint_matrix[feature_idx] 
                * probility_matrix[j+tidx_label*batch_size] * scalar;
        }
    }    
}

// version 1 nested loops slower
__device__ void d_matrixMatrixMultiply(
    FeatureType* datapoint_matrix,
    FeatureType* probility_matrix,
    float scalar,
    size_t batch_size,
    size_t num_features,
    size_t threads_per_mini_batch,
    FeatureType* result) {

    size_t tidx = threadIdx.x;
    size_t bidx = blockIdx.x;


    for(int m = 0 ; m < LABEL_CLASS ; m++){
        for (int j = 0; j < batch_size; j++) {
            for (int i = tidx; i < num_features; i += threads_per_mini_batch) {
                // index of the point with respect to the whole dataset
                size_t point_idx = bidx * batch_size + j;
                // index of the feature with respect to all features in the dataset
                size_t feature_idx = point_idx * num_features + i;
                //gradient result 
                result[i+m*num_features] += datapoint_matrix[feature_idx] 
                    * probility_matrix[j+m*batch_size] * scalar;
            }
        }
    }
}


//parallel implemetation of matrixTranspose
// v1 use tidx 0 - tidx 10 every point
__device__ void d_matrixTranspose(
    FeatureType* probility_matrix,
    FeatureType* probility_transpose,
    size_t batch_size,
    size_t relative_tidx,
    size_t point_idx_in_block){
    
    //transpose from batch * Label to Label * batch
    if(relative_tidx < LABEL_CLASS){

        probility_transpose[relative_tidx*batch_size+point_idx_in_block] =
             probility_matrix[relative_tidx+point_idx_in_block*LABEL_CLASS];

    }
}

//parallel implementation of matrix transpose
// different from v1 use tidx 0 - threads_per_mini_batch
__device__ void d_matrixTranspose2(
    FeatureType* probility_matrix,
    FeatureType* probility_transpose,
    size_t batch_size){

    // size_t threads_per_datapoint = threads_per_mini_batch / batch_size;
    // size_t relative_tidx = threadIdx.x % threads_per_datapoint;
    
    
    //transpose from batch * Label to Label * batch
    if(threadIdx.x < (LABEL_CLASS * batch_size)){
        //calcuate the first  LABEL_CLASS * batch_size thread
        // idx relative to the label
        size_t tidx_label = threadIdx.x / LABEL_CLASS;
        size_t relative_tidx_label = threadIdx.x % LABEL_CLASS;

        probility_transpose[relative_tidx_label*batch_size+tidx_label] =
             probility_matrix[relative_tidx_label+tidx_label*LABEL_CLASS];

    }
}


// updates the parameters using atomics
__device__ void d_updateParameters(
    FeatureType* gradient,
    FeatureType* parameter_vector,
    size_t num_features,
    size_t threads_per_mini_batch,
    double step_size) {

    size_t tidx = threadIdx.x;
    
    for (size_t i = tidx; i < num_features * LABEL_CLASS; i += threads_per_mini_batch) {
        FeatureType gradient_times_step_size = gradient[i] * step_size;
        atomicAdd(&parameter_vector[i], -gradient_times_step_size);
    }

}

// posibilily another way..
// speed almost the same 
//  __device__ void d_updateParameters(
//     FeatureType* gradient,
//     FeatureType* parameter_vector,
//     size_t num_features,
//     size_t threads_per_mini_batch,
//     double step_size) {

//     // printf("enter update parameters in sgd_single_point\n");

//     // size_t thread_offset = threadIdx.x % threads_per_datapoint;


//     for(size_t i= 0;i<LABEL_CLASS;i++){
      
//         for (size_t j = threadIdx.x; j < num_features; j+=threads_per_mini_batch){
  
//             atomicAdd(&parameter_vector[j+i*num_features], -gradient[j+i*num_features]*step_size);
//         }        
//     }        
// }  

// initializes all values in array to a certain value
__device__ void d_memset(
    FeatureType* array,
    float value,
    size_t num_elements,
    size_t threads_per_mini_batch) {

    size_t tidx = threadIdx.x;
    for (size_t i = tidx; i < num_elements; i += threads_per_mini_batch) {
        array[i] = value;
    }
}


// computes logistic function for a given parameter vector (theta) and a data point (x_i)
// double p_logisticFunction(FeatureType* d_theta, FeatureType* d_x_i, const size_t num_features) {
//      return logisticFunction(p_dotProduct(d_theta, d_x_i, num_features));
// }

// void p_softmaxFunction(FeatureType* d_theta, 
//     FeatureType* d_x_i, 
//     const size_t num_features,
//     FeatureType* result){
        

// }


// double logisticFunction(FeatureType* d_theta, FeatureType* d_x_i, const size_t num_features) {
    // return d_logisticFunction(dotProduct(d_theta, d_x_i, num_features));
//}


// computes logistic function with fast exp
__device__ float d_logisticFunction(float exponent) {
    return (1.0f / (1.0f + __expf(-exponent)));
}



// one - way dimention parallel softmax function
__device__ void d_softMaxFunction1(FeatureType* shared_memory, 
    FeatureType* posibility_each,
    size_t point_idx_in_shmem,
    size_t relative_tidx,
    size_t point_idx_in_block,
    size_t num_label) {
    //copy (theta)T x and take fast exponential
    if(relative_tidx < num_label){
        posibility_each[point_idx_in_block * num_label+relative_tidx]
            = __expf(shared_memory[relative_tidx * blockDim.x+ point_idx_in_shmem]);
    }
    __syncthreads();

    //calculate sum , each thread has a copy (++)
    float sum = 0;
    for (size_t i=0;i<num_label;i++){
        sum += posibility_each[point_idx_in_block * num_label + i];
    }
    __syncthreads();
    
    //calculate final posibility for each point
    if(relative_tidx < num_label){
        posibility_each[point_idx_in_block * num_label+relative_tidx]/=sum;
    }
    __syncthreads();
}

//two - way dimention parallel softmax function
__device__ void d_softMaxFunction2(FeatureType* shared_memory, 
    FeatureType* posibility_each,
    size_t point_idx_in_shmem,
    size_t relative_tidx,
    size_t point_idx_in_block,
    size_t num_thread_each_label) {
    //copy (theta)T x and take fast exponential
    if(relative_tidx < LABEL_CLASS){
        posibility_each[point_idx_in_block * LABEL_CLASS+relative_tidx]
            = __expf(shared_memory[relative_tidx * num_thread_each_label + point_idx_in_shmem]);
    }
    __syncthreads();

    //calculate sum , each thread has a copy (++)
    float sum = 0;
    for (size_t i=0;i<LABEL_CLASS;i++){
        sum += posibility_each[point_idx_in_block * LABEL_CLASS + i];
    }
    __syncthreads();
    
    //calculate final posibility for each point
    if(relative_tidx < LABEL_CLASS){
        posibility_each[point_idx_in_block * LABEL_CLASS+relative_tidx]/=sum;
    }
    __syncthreads();
}

// version 1 softmaxFunction for mbgd - 2
__device__ void d_softMaxFunction3(FeatureType* shared_memory, 
    FeatureType* posibility_each,
    size_t relative_tidx,
    size_t point_idx_in_batch,
    size_t num_label) {
    //copy (theta)T x and take fast exponential
    if(relative_tidx < num_label){
        posibility_each[point_idx_in_batch * num_label+relative_tidx]
            = __expf(shared_memory[relative_tidx * blockDim.x]);
    }
    __syncthreads();

    //calculate sum , each thread has a copy (++)
    float sum = 0;
    for (size_t i=0;i<num_label;i++){
        sum += posibility_each[point_idx_in_batch * num_label + i];
    }
    __syncthreads();
    
    //calculate final posibility for each point
    if(relative_tidx < num_label){
        posibility_each[point_idx_in_batch * num_label+relative_tidx]/=sum;
    }
    __syncthreads();
}

// version 4 softmax function for mbgd - 2 
__device__ void d_softMaxFunction4(FeatureType* shared_memory, 
    FeatureType* posibility_each,
    size_t relative_tidx,
    size_t point_idx_in_batch,
    size_t num_thread_each_label) {
    //copy (theta)T x and take fast exponential
    if(relative_tidx < LABEL_CLASS){
        posibility_each[point_idx_in_batch * LABEL_CLASS+relative_tidx]
            = __expf(shared_memory[relative_tidx * num_thread_each_label]);
    }
    __syncthreads();

    //calculate sum , each thread has a copy (++)
    float sum = 0;
    for (size_t i=0;i<LABEL_CLASS;i++){
        sum += posibility_each[point_idx_in_batch * LABEL_CLASS + i];
    }
    __syncthreads();
    
    //calculate final posibility for each point
    if(relative_tidx < LABEL_CLASS){
        posibility_each[point_idx_in_batch * LABEL_CLASS+relative_tidx]/=sum;
    }
    __syncthreads();
}

// verify the device properties satisfy the assumptions of the kernel
// check that the resulting grid and block dimensions
// dont' violate device limits
bool checkDeviceProps(
    size_t shared_memory_size,
    dim3 block_size,
    dim3 grid_size) {

    bool devicePropsOK = true;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    if (shared_memory_size > deviceProp.sharedMemPerBlock) {

        // printf("Shared Memory size exceeds maximum allowed size.\n");
        printf("SM-");
        devicePropsOK = false;
    }

    if (block_size.x > deviceProp.maxThreadsDim[0]
        || grid_size.x > deviceProp.maxGridSize[0]) {

        // printf("Grid or block size exceeds maximum allowed size.\n");
        printf("B-");
        devicePropsOK = false;
    }

    return devicePropsOK;
}

// updates the parameters (theta)
// void p_updateParameters(FeatureType* d_theta, FeatureType* d_gradient, size_t num_features, float step_size, bool revert) {
     // float sign = revert ? 1 : -1;
     // step_size *= sign;
     // addVectors(d_theta, d_gradient, num_features, step_size);
     // hipblasSaxpy(handle, num_features, &step_size, d_gradient, 1, d_theta, 1);
//}
